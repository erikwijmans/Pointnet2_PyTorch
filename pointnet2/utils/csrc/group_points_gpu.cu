#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"
#include "group_points_gpu.h"

// input: points(b, c, n) idx(b, npoints, nsample)
// output: out(b, c, npoints, nsample)
__global__ void group_points_kernel(int b, int c, int n, int npoints,
				    int nsample,
				    const float *__restrict__ points,
				    const int *__restrict__ idx,
				    float *__restrict__ out) {
    int batch_index = blockIdx.x;
    points += batch_index * n * c;
    idx += batch_index * npoints * nsample;
    out += batch_index * npoints * nsample * c;

    const int index = threadIdx.y * blockDim.x + threadIdx.x;
    const int stride = blockDim.y * blockDim.x;
    for (int i = index; i < c * npoints; i += stride) {
	const int l = i / npoints;
	const int j = i % npoints;
	for (int k = 0; k < nsample; ++k) {
	    int ii = idx[j * nsample + k];
	    out[(l * npoints + j) * nsample + k] = points[l * n + ii];
	}
    }
}

void group_points_kernel_wrapper(int b, int c, int n, int npoints, int nsample,
				 const float *points, const int *idx,
				 float *out, hipStream_t stream) {

    hipError_t err;
    group_points_kernel<<<b, opt_block_config(npoints, c), 0, stream>>>(
	b, c, n, npoints, nsample, points, idx, out);

    err = hipGetLastError();
    if (hipSuccess != err) {
	fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
	exit(-1);
    }
}

// input: grad_out(b, c, npoints, nsample), idx(b, npoints, nsample)
// output: grad_points(b, c, n)
__global__ void group_points_grad_kernel(int b, int c, int n, int npoints,
					 int nsample,
					 const float *__restrict__ grad_out,
					 const int *__restrict__ idx,
					 float *__restrict__ grad_points) {
    int batch_index = blockIdx.x;
    grad_out += batch_index * npoints * nsample * c;
    idx += batch_index * npoints * nsample;
    grad_points += batch_index * n * c;

    const int index = threadIdx.y * blockDim.x + threadIdx.x;
    const int stride = blockDim.y * blockDim.x;
    for (int i = index; i < c * npoints; i += stride) {
	const int l = i / npoints;
	const int j = i % npoints;
	for (int k = 0; k < nsample; ++k) {
	    int ii = idx[j * nsample + k];
	    atomicAdd(grad_points + l * n + ii,
		      grad_out[(l * npoints + j) * nsample + k]);
	}
    }
}

void group_points_grad_kernel_wrapper(int b, int c, int n, int npoints,
				      int nsample, const float *grad_out,
				      const int *idx, float *grad_points,
				      hipStream_t stream) {
    hipError_t err;
    group_points_grad_kernel<<<b, opt_block_config(npoints, c), 0, stream>>>(
	b, c, n, npoints, nsample, grad_out, idx, grad_points);

    err = hipGetLastError();
    if (hipSuccess != err) {
	fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
	exit(-1);
    }
}
