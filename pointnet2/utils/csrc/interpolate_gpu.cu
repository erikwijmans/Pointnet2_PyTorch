#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"
#include "interpolate_gpu.h"

// input: unknown(b, n, 3) known(b, m, 3)
// output: dist2(b, n, 3), idx(b, n, 3)
__global__ void three_nn_kernel(int b, int n, int m,
				const float *__restrict__ unknown,
				const float *__restrict__ known,
				float *__restrict__ dist2,
				int *__restrict__ idx) {
    int batch_index = blockIdx.x;
    unknown += batch_index * n * 3;
    known += batch_index * m * 3;
    dist2 += batch_index * n * 3;
    idx += batch_index * n * 3;

    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int j = index; j < n; j += stride) {
	float ux = unknown[j * 3 + 0];
	float uy = unknown[j * 3 + 1];
	float uz = unknown[j * 3 + 2];

	double best1 = 1e40, best2 = 1e40, best3 = 1e40;
	int besti1 = 0, besti2 = 0, besti3 = 0;
	for (int k = 0; k < m; ++k) {
	    float x = known[k * 3 + 0];
	    float y = known[k * 3 + 1];
	    float z = known[k * 3 + 2];
	    float d =
		(ux - x) * (ux - x) + (uy - y) * (uy - y) + (uz - z) * (uz - z);
	    if (d < best1) {
		best3 = best2;
		besti3 = besti2;
		best2 = best1;
		besti2 = besti1;
		best1 = d;
		besti1 = k;
	    } else if (d < best2) {
		best3 = best2;
		besti3 = besti2;
		best2 = d;
		besti2 = k;
	    } else if (d < best3) {
		best3 = d;
		besti3 = k;
	    }
	}
	dist2[j * 3 + 0] = best1;
	dist2[j * 3 + 1] = best2;
	dist2[j * 3 + 2] = best3;

	idx[j * 3 + 0] = besti1;
	idx[j * 3 + 1] = besti2;
	idx[j * 3 + 2] = besti3;
    }
}

void three_nn_kernel_wrapper(int b, int n, int m, const float *unknown,
			     const float *known, float *dist2, int *idx,
			     hipStream_t stream) {

    hipError_t err;
    three_nn_kernel<<<b, opt_n_threads(n), 0, stream>>>(b, n, m, unknown, known,
							dist2, idx);

    err = hipGetLastError();
    if (hipSuccess != err) {
	fprintf(stderr, "CUDA kernel "
			"failed : %s\n",
		hipGetErrorString(err));
	exit(-1);
    }
}

// input: points(b, c, m), idx(b, n, 3), weight(b, n, 3)
// output: out(b, c, n)
__global__ void three_interpolate_kernel(int b, int c, int m, int n,
					 const float *__restrict__ points,
					 const int *__restrict__ idx,
					 const float *__restrict__ weight,
					 float *__restrict__ out) {
    int batch_index = blockIdx.x;
    points += batch_index * m * c;

    idx += batch_index * n * 3;
    weight += batch_index * n * 3;

    out += batch_index * n * c;

    const int index = threadIdx.y * blockDim.x + threadIdx.x;
    const int stride = blockDim.y * blockDim.x;
    for (int i = index; i < c * n; i += stride) {
	const int l = i / n;
	const int j = i % n;
	float w1 = weight[j * 3 + 0];
	float w2 = weight[j * 3 + 1];
	float w3 = weight[j * 3 + 2];

	int i1 = idx[j * 3 + 0];
	int i2 = idx[j * 3 + 1];
	int i3 = idx[j * 3 + 2];

	out[i] = points[l * m + i1] * w1 + points[l * m + i2] * w2 +
		 points[l * m + i3] * w3;
    }
}

void three_interpolate_kernel_wrapper(int b, int c, int m, int n,
				      const float *points, const int *idx,
				      const float *weight, float *out,
				      hipStream_t stream) {

    hipError_t err;
    three_interpolate_kernel<<<b, opt_block_config(n, c), 0, stream>>>(
	b, c, m, n, points, idx, weight, out);

    err = hipGetLastError();
    if (hipSuccess != err) {
	fprintf(stderr, "CUDA kernel "
			"failed : %s\n",
		hipGetErrorString(err));
	exit(-1);
    }
}

// input: grad_out(b, c, n), idx(b, n, 3), weight(b, n, 3)
// output: grad_points(b, c, m)

__global__ void three_interpolate_grad_kernel(
    int b, int c, int n, int m, const float *__restrict__ grad_out,
    const int *__restrict__ idx, const float *__restrict__ weight,
    float *__restrict__ grad_points) {
    int batch_index = blockIdx.x;
    grad_out += batch_index * n * c;
    idx += batch_index * n * 3;
    weight += batch_index * n * 3;
    grad_points += batch_index * m * c;

    const int index = threadIdx.y * blockDim.x + threadIdx.x;
    const int stride = blockDim.y * blockDim.x;
    for (int i = index; i < c * n; i += stride) {
	const int l = i / n;
	const int j = i % n;
	float w1 = weight[j * 3 + 0];
	float w2 = weight[j * 3 + 1];
	float w3 = weight[j * 3 + 2];

	int i1 = idx[j * 3 + 0];
	int i2 = idx[j * 3 + 1];
	int i3 = idx[j * 3 + 2];

	atomicAdd(grad_points + l * m + i1, grad_out[i] * w1);
	atomicAdd(grad_points + l * m + i2, grad_out[i] * w2);
	atomicAdd(grad_points + l * m + i3, grad_out[i] * w3);
    }
}

void three_interpolate_grad_kernel_wrapper(int b, int n, int c, int m,
					   const float *grad_out,
					   const int *idx, const float *weight,
					   float *grad_points,
					   hipStream_t stream) {

    hipError_t err;
    three_interpolate_grad_kernel<<<b, opt_block_config(n, c), 0, stream>>>(
	b, n, c, m, grad_out, idx, weight, grad_points);

    err = hipGetLastError();
    if (hipSuccess != err) {
	fprintf(stderr, "CUDA kernel "
			"failed : %s\n",
		hipGetErrorString(err));
	exit(-1);
    }
}
